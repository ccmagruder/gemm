
#include <hip/hip_runtime.h>
#include <stdexcept>

void cudaCheck(hipError_t code, const char* file, const int line) {
    if (code != hipSuccess) {
        char msg[100];
        sprintf(msg, "GPU kernel assert: %s:%d \"%s\"\n", file, line,
                hipGetErrorString(code));
        throw std::runtime_error(msg);
    }
}

void cudaCheck(const char* file, const int line) {
    hipError_t code = hipGetLastError();
    cudaCheck(code, file, line);
}
