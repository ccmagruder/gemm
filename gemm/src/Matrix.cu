#include "Matrix.h"
#include "utils.cuh"

void Matrix::DevDeleter::operator()(float* devPtr) {
    if (devPtr) {
        hipFree(devPtr);
        devPtr = nullptr;
    }
}

void Matrix::_devAlloc() const {
    if (!this->_dev_ptr) {
        float* ptr;
        hipError_t code =
            hipMalloc((void**)&ptr, this->m * this->n * sizeof(float));
        cudaCheck(code, __FILE__, __LINE__);
        this->_dev_ptr = std::unique_ptr<float, DevDeleter>(ptr);
    }
}

void Matrix::toDevice() const {
    this->_devAlloc();
    hipError_t code =
        hipMemcpy(this->_dev_ptr.get(), this->_host_ptr.get(),
                   this->m * this->n * sizeof(float), hipMemcpyHostToDevice);
    cudaCheck(code, __FILE__, __LINE__);
}

void Matrix::toHost() {
    hipError_t code =
        hipMemcpy(this->_host_ptr.get(), this->_dev_ptr.get(),
                   this->m * this->n * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheck(code, __FILE__, __LINE__);
}
