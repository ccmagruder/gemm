#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void __passthrough() {
    extern float* smem[];
}

// TODO: move to kernels.cu
void setMaxSharedMemory(void (*kernel)(void)) {
    int device;
    int sharedMemoryPerBlockOptin;
    // TODO: Refactor checkCuda(code, FILE, LINE)
    // hipError_t code;

    hipGetDevice(&device);
    hipDeviceGetAttribute(&sharedMemoryPerBlockOptin,
                           hipDeviceAttributeSharedMemPerBlockOptin, device);
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
                         sharedMemoryPerBlockOptin);
}

void sharedMemoryAlloc() {
    float* ptr;
    hipError_t code = hipMalloc((void**)&ptr, sizeof(float));
    setMaxSharedMemory(__passthrough);
    __passthrough<<<1, 1, 99 * 1024>>>();
    cudaCheck(__FILE__, __LINE__);
    hipFree(ptr);
}
